#include "hip/hip_runtime.h"
#include "poissonSample.h"
#include "TimerClock.hpp"

#pragma comment(lib, "hiprand.lib")
static int max_level = 7;
static float d = 0.05f;


struct offset
{
	offset(int _x, int _y, int _z) {
		x = _x;
		y = _y;
		z = _z;
		dist = _x * _x + _y * _y + _z * _z;
	}
	int x;
	int y;
	int z;
	int dist;
};
bool compare_offset(const offset& offset1, const offset& offset2)
{
	return offset1.dist < offset2.dist;
}

int PGCompute(int x, int y, int z)
{
	x = x % 3 ? x % 3 : 3;
	y = y % 3 ? y % 3 : 3;
	z = z % 3 ? z % 3 : 3;

	return x + (y - 1) * 3 + (z - 1) * 3 * 3;
}

//cpu random generator
inline double UniformRandom()
{
	return static_cast<double>(rand()) / RAND_MAX;
}

//gpu random generator
struct psrngen
{
	__host__ __device__ psrngen(float _a, float _b) : a(_a), b(_b) { ; }

	__host__ __device__ float operator()(const unsigned int n) const
	{
		thrust::default_random_engine rng;
		thrust::uniform_real_distribution<float> dist(a, b);
		rng.discard(n);
		return dist(rng);
	}
	float a, b;

};

//filterPoissonSample���Ӻ��������ڽ�����������Ĺ���/���ȹ��͵�sample����
//a sub-function of filterPoissonSample. Leave out the samples close to the neuron soma/the samples with low intensity.
__global__ 
void filterPoissonSampleKernel(int sampleNum, dim3 center, int centerRadius, 
	int poissonSampleThreshold, int width, int height, int slice, uchar* d_imagePtr, uchar* d_isSampleValid, int* d_seedPos, float* d_X_new_out, float* d_Y_new_out, float* d_Z_new_out)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= sampleNum) return;
	float x, y, z;
	int xInt, yInt, zInt;
	x = d_X_new_out[idx];
	if (x == 0) return;
	y = d_Y_new_out[idx];
	z = d_Z_new_out[idx];

	xInt = x * (width - 1) + 0.5f;
	yInt = y * (height - 1) + 0.5f;
	zInt = z * (slice - 1) + 0.5f;

	if ((xInt - center.x) * (xInt - center.x) + (yInt - center.y) * (yInt - center.y) + (zInt - center.z) * (zInt - center.z)
		< centerRadius * centerRadius * 16) return;

	if (d_imagePtr[zInt * width * height + yInt * width + xInt] > poissonSampleThreshold)
	{
		d_isSampleValid[idx] = 1;
		d_seedPos[idx] = zInt * width * height + yInt * width + xInt;
	}
}

/*
������filterPoissonSample_gpu
���ܣ�������������Ĺ���/���ȹ��͵�sample����
���룺d_X/Y/Z�� ��gpu���������ӵ��������飻center����������λ�ã�centerRadius������뾶
�����seedArr(���ɸѡ������ӵ��±�)
*/
/*
Function��filterPoissonSample_gpu
Work��Filter the random generated samples.The samples close to the center of neuron soma/the samples of low intensity are dropped.
Input��d_X/Y/Z�� The coordinate of the seeds; center��the location of neuron center; centerRadius��the control radius of neuron center
Output��seedArr (the indices of filtered seeds in the original image)
*/

void filterPoissonSample_gpu(std::vector<int>& seedArr, dim3 center, int centerRadius, int sampleNum, 
	int width, int height, int slice, uchar* d_imagePtr, float* d_X_new_out, float* d_Y_new_out, float* d_Z_new_out)
{
	//����Сkernel�Ƿ���cudamallocmanaged()��һ�㣿
	int validCount = 0;

	uchar* isSampleValid = (uchar*)malloc(sizeof(uchar) * sampleNum);
	memset(isSampleValid, 0, sizeof(uchar) * sampleNum);

	uchar* d_isSampleValid;
	hipMalloc(&d_isSampleValid, sizeof(uchar) * sampleNum);
	hipMemset(d_isSampleValid, 0, sizeof(uchar) * sampleNum);
	uchar poissonSampleThreshold = 10;
	poissonSampleThreshold = 1;

	int* seedPos = (int*)malloc(sizeof(int) * sampleNum);
	int* d_seedPos;
	hipMalloc(&d_seedPos, sizeof(int) * sampleNum);



	//������������Ĺ���/���ȹ��͵�sample����
	//filter the samples
	filterPoissonSampleKernel << < (sampleNum - 1) / 256 + 1, 256 >> > (sampleNum, center, centerRadius, poissonSampleThreshold,
		width, height, slice, d_imagePtr, d_isSampleValid, d_seedPos, d_X_new_out, d_Y_new_out, d_Z_new_out);
	
	//����ʣ�����Ч���Ӹ���
	//calculating the number of valid samples
	validCount = thrust::reduce(thrust::device, d_isSampleValid, d_isSampleValid + sampleNum, 0, thrust::plus<int>());

	//ʣ���������̫���ˣ��ʵ��������������ֵ(Ҳ����ɾ���˽׶�)
	//if the number of valid samples are too much, increase the threshold. (This process can be removed.)
	while (validCount > 500 && poissonSampleThreshold <= 250)
	{
		validCount = 0;
		hipMemset(d_isSampleValid, 0, sizeof(uchar) * sampleNum);
		poissonSampleThreshold += 5;

		filterPoissonSampleKernel << < (sampleNum - 1) / 256 + 1, 256 >> > (sampleNum, center, centerRadius, poissonSampleThreshold,
			width, height, slice, d_imagePtr, d_isSampleValid, d_seedPos, d_X_new_out, d_Y_new_out, d_Z_new_out);
		validCount = thrust::reduce(thrust::device, d_isSampleValid, d_isSampleValid + sampleNum, 0, thrust::plus<int>());

		printf("poissonSampleThreshold: %d, validCount: %d\n", poissonSampleThreshold, validCount);
	}

	hipMemcpy(isSampleValid, d_isSampleValid, sizeof(uchar) * sampleNum, hipMemcpyDeviceToHost);
	hipMemcpy(seedPos, d_seedPos, sizeof(int) * sampleNum, hipMemcpyDeviceToHost);
	
	for (int i = 0; i < sampleNum; i++)
	{
		if (isSampleValid[i])
		{
			seedArr.push_back(seedPos[i]);
		}
	}


	//������������Ϊ���һ������
	//Push the neuron center as the last seed point
	seedArr.push_back(center.z * width * height + center.y * width + center.x);
	std::cerr << "center idx: " << center.z * width * height + center.y * width + center.x << std::endl;
	std::cerr << "center_x: " << center.x << " center_y: " << center.y << " center_z: " << center.z << std::endl;
	validCount++;

	std::cerr << "Total num of samples GPU:" << sampleNum << std::endl;
	std::cerr << "Total num of valid samples GPU:" << validCount << std::endl;

	hipFree(d_isSampleValid);
	hipFree(d_seedPos);
	free(seedPos);
	free(isSampleValid);
}



//�������㲿��
//Poisson sampling kernel
__global__
void samplingKernel2(float* d_X_new, float* d_Y_new, float* d_Z_new,
	float* d_X_new_out, float* d_Y_new_out, float* d_Z_new_out,
	float* d_Random,
	int curSize, float r,
	int PG, int level,
	int* d_offset_x, int* d_offset_y, int* d_offset_z,
	uchar* d_imagePtr, uchar* d_visited, int* d_compress, int width, int height, int slice, int seedValueThreshold)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if (idx >= curSize) return; //curSize�����ǵ�ǰPG�ĳ��ȱ��� level=2��PG=1, curSize = 4
	//���threadidx,���ֵֻ���ǵ�ǰPhase�ĸ�����������磺level=2, PG=1, ����Ϊ4
	//d_X_new�������4����㣬��ôthreadIdx����1-4��ÿ��ȡһ�����
	//printf("\n\nidx:%d\n", idx);
	//printf("cursize:%d\n", curSize);

	float curX = d_X_new[idx];
	float curY = d_Y_new[idx];
	float curZ = d_Z_new[idx];
	//printf("%d %f %f %f\n", idx, curX, curY, curZ);
	//printf("%f %f\n", curX, curY);
	//curX,curY�ǵ�ǰ�������ꡣͨ�����������Լ��㵱ǰ����index
	//���˶�ά�ĵ�ǰ���index,����offset���Ϳ��Լ���neighbor���Ķ�άindex.
	//ͨ��neighbor����һάindex�����Եõ�neighbor������sample�����ꡣ����d_X_new_out[neighhborIndex1d]
	//��ά���������(0,0),(3,0),(0,3),(3,3), һά���������0, 3, 12, 15

	//���񳤶�r
	int3 curIndex3d = { -1,-1,-1 };
	for (float t = 0; t < curX; t += r, curIndex3d.x += 1);
	for (float t = 0; t < curY; t += r, curIndex3d.y += 1);
	for (float t = 0; t < curZ; t += r, curIndex3d.z += 1);
	int ny = powf(2, level) + 0.5; //nx,ny,nz:ÿ��ά���ϵ��������
	int nx = ny;
	int nz = nx;
	int curIndex1d = curIndex3d.z * ny * nx + curIndex3d.y * nx + curIndex3d.x;
	//printf("%f %f %d %d %d\n", curX, curY, curIndex2d.x, curIndex2d.y, curIndex1d);
	//printf("d_X_new_out[curIndex1d] : %f\n", d_X_new_out[curIndex1d]);

	float r_x = width * 1.f / nx;
	float r_y = height * 1.f / ny;
	float r_z = slice * 1.f / nz; //ע��, ����<1�������㹻�����Ժ�> 1

	//printf("rx: %.2f, ry: %.2f, rz: %.2f\n", r_x, r_y, r_z);

	//�� ��PG�е����
	{
		if (d_X_new_out[curIndex1d] < 1e-5) //���������ǿյ�(=0)
		{
			//d_random()��[0,1]ȡֵ��-0.5������[-0.5,0.5]
			/*d_X_new_out[curIndex1d] = d_X_new[idx] + (d_Random[curIndex1d] - 0.5)*r;
			d_Y_new_out[curIndex1d] = d_Y_new[idx] + (d_Random[curIndex1d + 1] - 0.5)*r;
			d_Z_new_out[curIndex1d] = d_Z_new[idx] + (d_Random[curIndex1d + 2] - 0.5)*r;*/
			//r:��ǰ���񳤶ȡ�ʵ���������Ͷ��Ҳ���������������ڲ�Ͷ�㡣
			//��Ϊd_X_new[idx]���������ĵ����꣬�����ĳ���x,y,zƫ��0.5r��û�г�����

			bool find = false;

			int zstart = MAX(0, curIndex3d.z * r_z);
			int ystart = MAX(0, curIndex3d.y * r_y);
			int xstart = MAX(0, curIndex3d.x * r_x);

			int zend = MIN(slice, (curIndex3d.z + 1) * r_z);
			int yend = MIN(height, (curIndex3d.y + 1) * r_y);
			int xend = MIN(width, (curIndex3d.x + 1) * r_x);

			int tz, ty, tx;
			for (tz = zstart; tz < zend && find == false; tz++)
				for (ty = ystart; ty < yend && find == false; ty++)
					for (tx = xstart; tx < xend && find == false; tx++)
					{
						int fullIdx = tz * width * height + ty * width + tx;
						if (d_imagePtr[fullIdx] >= seedValueThreshold && d_visited[fullIdx] == 0)
						{
							find = true;
							d_X_new_out[curIndex1d] = tx *1.0f / width;
							d_Y_new_out[curIndex1d] = ty *1.0f / height;
							d_Z_new_out[curIndex1d] = tz *1.0f / slice;
							d_visited[fullIdx] = 1;
						}
					}

			//printf("idx: %d, tz: %d %d, ty: %d %d, tx: %d %d\n", idx, zstart, zend, ystart, yend, xstart, xend);

			/*if (find)
			{
				printf("find!, idx: %d\n", idx);
			}*/

		}
		else//����Ѿ�Ͷ���ˣ����������
		{
			return;
		}
	}


	//ͬ��
	__syncthreads();
	//����Ŀǰ PG ��֮ǰ���� PG �ڵĵ�

	for (int i = 0; i < 93; i++)
	{
		int3 offset;
		offset.x = d_offset_x[i];
		offset.y = d_offset_y[i];
		offset.z = d_offset_z[i];
		if (offset.x == 0 && offset.y == 0 && offset.z == 0) continue;

		int3 neighborIndex3d;
		neighborIndex3d.x = offset.x + curIndex3d.x;
		neighborIndex3d.y = offset.y + curIndex3d.y;
		neighborIndex3d.z = offset.z + curIndex3d.z;
		//printf("nx, ny, neiboridx: %d %d %d %d\n",nx, ny, neighborIndex2d.x, neighborIndex2d.y);
		if (neighborIndex3d.x < 0 || neighborIndex3d.x >= nx || neighborIndex3d.y < 0 || neighborIndex3d.y >= ny || neighborIndex3d.z < 0 || neighborIndex3d.z >= nz)
			continue;

		int neighborIndex1d = neighborIndex3d.x + nx * neighborIndex3d.y + nx * ny*neighborIndex3d.z;
		//printf("%d %d %d %d\n",curIndex2d.x, curIndex2d.y,  neighborIndex2d.x, neighborIndex2d.y);
		//neighborIndex ָ����neighbor����index, ��������sample��ʲô����neiborPos2d.

		//����index��Ӧ�����ʵ���꣬Ҳ����Pos
		float3 curPos3d;
		curPos3d.x = d_X_new_out[curIndex1d];
		curPos3d.y = d_Y_new_out[curIndex1d];
		curPos3d.z = d_Z_new_out[curIndex1d];
		float3 neighborPos3d;
		neighborPos3d.x = d_X_new_out[neighborIndex1d];
		neighborPos3d.y = d_Y_new_out[neighborIndex1d];
		neighborPos3d.z = d_Z_new_out[neighborIndex1d];
		//Ϊ0˵�����ھ��޵� ���ü�� ֱ��������һ��for loop����
		if (neighborPos3d.x == 0) continue;

		//����dist����r_1 r_2 �ж��Ƿ���ײ
		//dist�õ���neighbor��sample�����꣬���Ǹ��
		float dist = sqrtf(powf(curPos3d.x - neighborPos3d.x, 2) + powf(curPos3d.y - neighborPos3d.y, 2) + powf(curPos3d.z - neighborPos3d.z, 2));
		//����r_1 r_2
		//�����Ƚ�����
		//printf("%f");
		//printf("%f %f %f %d %d %f %f\n\n", r1, r2, dist, curIndex1d, neighborIndex1d, d_X_new_out[curIndex1d], d_X_new_out[neighborIndex1d]);

		//Modified by jifaley 20210618
		//impX,impY: ͨ����ǰsample������(0.12,0.65)��������importance field����ľ����±�(963*0.12,305*0.65)
		int impX = curPos3d.x * width;
		int impY = curPos3d.y * height;
		int impZ = curPos3d.z * slice;

		int curImpfullIdx = impZ * width * height + impY * width + impX;
		int curImpSmallIdx = d_compress[curImpfullIdx];
		
		int r1 = r;
		int r2 = r;


		//nimpX, nimpY: �����ھ�sample����������ھӵ�importance
		int nimpX = neighborPos3d.x * width;
		int nimpY = neighborPos3d.y * height;
		int nimpZ = neighborPos3d.z * slice;
		int neighborImpfullIdx = nimpZ * width * height + nimpY * width + nimpX;
		int neighborImpSmallIdx = d_compress[neighborImpfullIdx];


		if (dist < (r1 + r2)*1.0f / 2)
		{
			d_X_new_out[curIndex1d] = 0;
			d_Y_new_out[curIndex1d] = 0;
			d_Z_new_out[curIndex1d] = 0;
			//std::cerr << "r1: " << r1 << " r2: " << r2 << " dist: " << dist <<  ' ' << curIndex1d << ' ' << neighborIndex1d << ' ' << curPos2d.x << ' ' << curPos2d.y << ' ' << neighborPos2d.x << ' '
			//	<< neighborPos2d.y << std::endl;

			return;
		}
	}
	//printf("d_X_new_out2(%d): %f\n", idx, d_Y_new_out[curIndex1d]);

}


/*
������doPoissonSample2
���ܣ�ͨ��Poisson�����������ӵ�
���룺d_imagePtr(ԭͼ), d_imagePtr_compact(��ѹ�����ͼ)
�����seedArr(���ɸѡ������ӵ��±�)
˼·�����ɲ�������Wei et al.�� Parallel Poisson Disk Sampling ʵ�֣���δ���г���Ż�
L.-Y. Wei. Parallel poisson disk sampling. Acm Transactions On Graphics(tog), 27(3) : 1�C9, 2008.
���Ƚ������ռ仮��Ϊ��ͬ��Phase Group(PG)��ÿ��PG֮�以��Ӱ�졣
d Ϊ�����������������ռ仮�ֵĳ߶ȣ�dԽС����Խϸ
Poisson���ʣ����������֮��ľ��벻�ᳬ��ĳ����ֵ(��Ҳ�ǻ���PG��ԭ��֮һ����
*/
/*
Function��doPoissonSample2
Work��The implementaion of parallel poisson disk sampling, generate seed points for initial neuron tracing
Input��d_imagePtr(the original image), d_imagePtr_compact(the image after stream compaction)
Output��seedArr(the indices of the generated seeds in the original image)
Implementation��See L.-Y. Wei. Parallel poisson disk sampling. Acm Transactions On Graphics(tog), 27(3) : 1�C9, 2008.
This implemenation in our work are not fully optimized.
Firstly, the full space are divided in to different Phase Group(PG)s, different PGs are independent.
d: The hyper parameter, denotes the fineness of space division. A small "d" will make a fine-grained division.
Poisson property: Two random samples will have a distance larger than a fixed value.
*/

int doPoissonSample2(std::vector<int>& seedArr, dim3 center, int centerRadius, int width, int height, int slice, int newSize, uchar* d_imagePtr, uchar* d_imagePtr_compact, int* d_compress, int* d_decompress)
{
	TimerClock timer;
	timer.update();
	//����
	//ȫ����ʼ��Ϊd
	std::cout << "d = " << d << std::endl;
	float r1, r2;
	r1 = 0.1f;
	r2 = 0.1f;
	float sumValue = 0;
	float meanValue = 0;
	float value = 0;
	std::cerr << "GPU sampling malloc took me " << timer.getTimerMilliSec() << " milliseconds." << std::endl;
	timer.update();


	float radius_square = 9;
	const int radius_int = static_cast<int>(floor(sqrt(radius_square)));
	int dimension = 3;
	std::vector<offset> offset_set;
	for (int i = -radius_int; i <= radius_int; i++)
	{
		for (int j = -radius_int; j <= radius_int; j++)
		{
			for (int k = -radius_int; k <= radius_int; k++)
			{
				offset aa(k, j, i);
				if (aa.dist < radius_square)
				{
					offset_set.push_back(aa);
				}
			}
		}
	}
	//���ݾ���Բ�ĵ�Զ�����򡣺���ɸѡ��ʱ�����жϾ����Լ������neighbor�Ƿ��ͻ�����ж��ⲿ��
	//Generate the phase groups.
	std::sort(offset_set.begin(), offset_set.end(), compare_offset);
	int* offset_x = (int*)malloc(sizeof(int) * offset_set.size());
	int* offset_y = (int*)malloc(sizeof(int) * offset_set.size());
	int* offset_z = (int*)malloc(sizeof(int) * offset_set.size());

	for (int i = 0; i < offset_set.size(); i++)
	{
		offset_x[i] = offset_set[i].x;
		offset_y[i] = offset_set[i].y;
		offset_z[i] = offset_set[i].z;
	}


	//����������ÿ��level�ĸ��
	//level 0 ��ʼ��

	std::cerr << "max_level: " << max_level << std::endl;
	int num = 1;
	float r = 1;
	int nx = 1;
	int ny = 1;
	int nz = 1;
	float x0 = 1 / 2.0f;
	float y0 = 1 / 2.0f;
	float z0 = 1 / 2.0f;
	int phase0 = 1;

	//������X Y PG
	//PG == Phase Group���������򻮷ֳ�Ϊ��ͬ��Phase(����:1-9), ʹ����ͬPhase�ڵ�
	//����֮������㹻Զ���Ա㿪չ���в���Ͷ��

	/*
		�����ö�ά���������ӣ�����һ����4 * 4 = 16�����񣬻���Ϊ9��,����ÿһ���Phase���£�
		Use 2-d grid as example. Assume there is 4 * = 16 grid, we divide them into 9 groups. The phase of the groups are as follows:
		1--2--3--1
		|  |  |  |
		7--8--9--7
		|  |  |  |
		4--5--6--4
		|  |  |  |
		1--2--3--1
		�������ֳ��������񣬿��Ա�֤��������1/����2/����3....֮��ľ��붼���ڵ���3
		This division can assure that the blocks with the same phase number have a distance >= 3.
	*/

	std::vector<std::vector<float>> X;
	std::vector<std::vector<float>> Y;
	std::vector<std::vector<float>> Z;
	std::vector<std::vector<int>> PG;

	//store all the points of current level
	std::vector<float> one_X;
	std::vector<float> one_Y;
	std::vector<float> one_Z;
	std::vector<int> one_PG;
	//store each point via emplace_back func
	one_X.emplace_back(x0);
	one_Y.emplace_back(y0);
	one_Z.emplace_back(z0);
	one_PG.emplace_back(phase0);
	//store the points of current level into respective level
	X.emplace_back(one_X);
	Y.emplace_back(one_Y);
	Z.emplace_back(one_Z);
	PG.emplace_back(one_PG);


	//ÿ��level��һ��ѭ��  
	for (int level = 1; level < max_level; level++)
	{
		num = pow(8.0, level); //num��ÿ��level��������������������ά�ģ�level=1����8�����飬level=2����4*4*4=64������
		//num: the number of blocks in each level.
		r = 1 / pow(2.0, level); //r��������Ŀ��
		//r: the width of block.
		nx *= 2;
		ny *= 2;
		nz *= 2;
		x0 = 1 / 2.0f * r; //x0,y0,z0���������½��������������,ע����������Phase=1
		y0 = 1 / 2.0f * r; //x0,y0,z0 is the coordinate of the first block. Note that this block's Phase equals 1.
		z0 = 1 / 2.0f * r;
		float x, y, z;

		//clean the points remaining and store the new points
		std::vector<float>().swap(one_X);
		std::vector<float>().swap(one_Y);
		std::vector<float>().swap(one_Z);
		std::vector<int>().swap(one_PG);
		//generate & store each point via emplace_back func

		for (int k = 0; k < nz; k++)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					x = x0 + i * r;
					y = y0 + j * r;
					z = z0 + k * r;
					one_X.emplace_back(x);
					one_Y.emplace_back(y);
					one_Z.emplace_back(z);
					phase0 = PGCompute(i + 1, j + 1, k + 1);//�±��0��ʼ ��Ҫ+1���� 
					one_PG.emplace_back(phase0);
				}
			}
		}

		//store the points of current level
		X.emplace_back(one_X);
		Y.emplace_back(one_Y);
		Z.emplace_back(one_Z);
		PG.emplace_back(one_PG);
	}

	std::cerr << "GPU sampling postprocessing 2nd stage took me " << timer.getTimerMilliSec() << " milliseconds." << std::endl;
	timer.update();

	//����Ϊֹ���������������е�Phase Group. ���濪ʼ������
	//�������������ɵ�1�׶εĳ�ʼ�㣨1���㣩�����滮��Ϊ��ͬlevel��ÿ��level����һ��level�̳�һЩ�㣬Ȼ��ͶһЩ�µ㡣

	//Till now, all of the Phase Groups are created. The sampling starts from here.
	//Sampling: First, generate the point in level 0 (only one point in the space); And the sampling process are
	//divided into multi levels. Each level will inherit some points in the last level, and generate a few new points.





	//����Xsampled �� X һģһ��
	std::vector<std::vector<float>> Xsampled(X);
	std::vector<std::vector<float>> Ysampled(Y);
	std::vector<std::vector<float>> Zsampled(Z);
	//Ŀǰ�������еĸ�㶼����ǰ�������
	//Xsampled ȫ������Ϊ0
	for (int level = 0; level < max_level; level++)
	{
		fill(Xsampled[level].begin(), Xsampled[level].end(), 0);
		fill(Ysampled[level].begin(), Ysampled[level].end(), 0);
		fill(Zsampled[level].begin(), Zsampled[level].end(), 0);
	}

	//��Xsampled�� level 0 ��Ψһһ���� ����
	Xsampled[0][0] = 1.0 / 2 + +(UniformRandom() - 0.5)*r;
	Ysampled[0][0] = 1.0 / 2 + +(UniformRandom() - 0.5)*r;
	Zsampled[0][0] = 1.0 / 2 + +(UniformRandom() - 0.5)*r;

	//��Xת��ΪX_new ÿһ����һ��X_new
	//���vector���±���PG��Ŀǰ,һ����27��Phase Group(3*3*3)
	std::vector<std::vector<float>> X_new(27);
	std::vector<std::vector<float>> Y_new(27);
	std::vector<std::vector<float>> Z_new(27);
	std::vector<int> count(27, 0);

	std::cerr << "Before random took me " << timer.getTimerMilliSec() << " milliseconds." << std::endl;
	timer.update();

	hiprandGenerator_t gen;//����������� ֻ����һ�ξͺ�

	uchar* d_visited;
	hipMalloc(&d_visited, sizeof(uchar) * width * height * slice);
	hipMemset(d_visited, 0, sizeof(uchar) * width * height * slice);


	std::cerr << "random init took me " << timer.getTimerMilliSec() << " milliseconds." << std::endl;
	timer.update();
	//hipError_t errorCheck;
	int pg;

	float* d_X_new;
	float* d_X_new_out;
	float* d_Y_new;
	float* d_Y_new_out;
	float* d_Z_new;
	float* d_Z_new_out;
	float* d_Random;
	int* d_offset_x;
	int* d_offset_y;
	int* d_offset_z;

	//lenXYZ:��level������������
	//lenXYZ_Max:����level�����ܵ���������
	//lenXYZ:the number of blocks at the current level.
	//lenXYZ_Max:The max possible number of blocks.

	int lenXYZ_max = pow(pow(2, max_level - 1), 3) + 1;
	std::cerr << lenXYZ_max << std::endl;
	hipMalloc((void**)&d_X_new, sizeof(float)*lenXYZ_max);
	hipMalloc((void**)&d_Y_new, sizeof(float)*lenXYZ_max);
	hipMalloc((void**)&d_Z_new, sizeof(float)*lenXYZ_max);
	hipMalloc((void**)&d_X_new_out, sizeof(float)*lenXYZ_max);
	hipMalloc((void**)&d_Y_new_out, sizeof(float)*lenXYZ_max);
	hipMalloc((void**)&d_Z_new_out, sizeof(float)*lenXYZ_max);
	hipMalloc((void**)&d_Random, sizeof(float)*lenXYZ_max + 2);
	hipMalloc((void**)&d_offset_x, sizeof(int) * 93);
	hipMalloc((void**)&d_offset_y, sizeof(int) * 93);
	hipMalloc((void**)&d_offset_z, sizeof(int) * 93);

	std::cerr << "malloc took me " << timer.getTimerMilliSec() << " milliseconds." << std::endl;
	timer.update();

	hipMemcpy(d_offset_x, offset_x, sizeof(int) * 93, hipMemcpyHostToDevice);
	hipMemcpy(d_offset_y, offset_y, sizeof(int) * 93, hipMemcpyHostToDevice);
	hipMemcpy(d_offset_z, offset_z, sizeof(int) * 93, hipMemcpyHostToDevice);


	float* h_X_new = (float*)malloc(sizeof(float)*lenXYZ_max);
	float* h_Y_new = (float*)malloc(sizeof(float)*lenXYZ_max);
	float* h_Z_new = (float*)malloc(sizeof(float)*lenXYZ_max);

	std::cerr << "memcpy took me " << timer.getTimerMilliSec() << " milliseconds." << std::endl;
	timer.update();


	hipError_t errorCheck;



	//ѭ��������Ĳ��� ����������kernel �̳нṹ��
	//The core processing (including sampling and inheriting)
	for (int level = max_level -1; level < max_level; level++) {

		//���㱾level��X_new(PG���)
		for (int n = 0; n < 27; n++)
		{
			X_new[n].clear();
			Y_new[n].clear();
			Z_new[n].clear();
		}

		//X[level].size()�ǵ�ǰlevel�ĵ�����
		for (int k = 0; k < X[level].size(); k++)
		{
			float x = X[level][k];
			float y = Y[level][k];
			float z = Z[level][k];
			//����ÿ��level�����е��Ӧ��PG
			int pg = PG[level][k];
			float R = 1.0f / pow(2.0f, level);
			X_new[pg - 1].push_back(x);
			Y_new[pg - 1].push_back(y);
			Z_new[pg - 1].push_back(z);
			//X_new_out[pg - 1].emplace_back(0);
			//Y_new_out[pg - 1].emplace_back(0);
		}


		//cout << "test" << endl;
		//�̳� ����Xsampled[level - 1] �ѵ�Ͷ�� X_new_out ��û��Ĳ���Ϊ0 kernel����0�����ж��Ƿ�Ͷ�㣩
		//��һ�������� ʹ���Ӧ�̳е������λ��
		for (int n = 0; n < 27; n++)
			count[n] = 0;

		for (int n = 0; n < Xsampled[level - 1].size(); n++)
		{
			float x = Xsampled[level - 1][n];
			float x_tmp = x;
			if (x_tmp == 0) //˵���ϸ�level�˴�û��Ͷ��
			{
				continue;
			}

			float y = Ysampled[level - 1][n];
			float y_tmp = y;
			float z = Zsampled[level - 1][n];
			float z_tmp = z;
			int nx, ny, nz;
			nx = ny = nz = (int)(pow(2.0f, level) + 0.5f);
			float blockSize = 1.0f / nx;
			//�����ϸ�level�����е��ڵ�ǰlevel��Ӧ��PG
			int ix = -1;
			int iy = -1;
			int iz = -1;
			while (x_tmp > 0)
			{
				ix++;
				x_tmp -= blockSize;
			}
			while (y_tmp > 0)
			{
				iy++;
				y_tmp -= blockSize;
			}
			while (z_tmp > 0)
			{
				iz++;
				z_tmp -= blockSize;
			}
			//index���Ǹõ��ڵ�ǰlevel���±�
			int index = ix + iy * nx + iz * nx * ny;

			pg = PG[level][index] - 1;
			Xsampled[level][index] = x;
			Ysampled[level][index] = y;
			Zsampled[level][index] = z;
			count[pg]++;
		}

		//9�� kernel parallel �����ÿ��PG��Ӧ�����в��������� ��� posXnew
		//��PG��ѭ��

		//��ǰ����device�ռ�
		int lenXYZ = 0;
		int curlenXYZ = 0;
		for (int PG = 0; PG < 27; PG++)
		{
			lenXYZ += X_new[PG].size();
		}

		//std::cerr << "Level: " << level << "lenXYZ: " << lenXYZ << std::endl;
		//����ĳ�lenXY = nx *ny Ӧ����һ���� 16 = 4* 4  = 4 + 1 +1 + ....
		//���ռ�

		//��ָ�붨λÿһ�οռ�
		float* d_cur_X_new = d_X_new;
		float* d_cur_Y_new = d_Y_new;
		float* d_cur_Z_new = d_Z_new;
		//float* d_cur_Random = d_Random;

		//hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);
		//���� level PG �������������

		hipMemcpy(d_X_new_out, &Xsampled[level][0], sizeof(float)*lenXYZ, hipMemcpyHostToDevice);
		hipMemcpy(d_Y_new_out, &Ysampled[level][0], sizeof(float)*lenXYZ, hipMemcpyHostToDevice);
		hipMemcpy(d_Z_new_out, &Zsampled[level][0], sizeof(float)*lenXYZ, hipMemcpyHostToDevice);

		float * h_cur_X_new = h_X_new;
		float * h_cur_Y_new = h_Y_new;
		float * h_cur_Z_new = h_Z_new;

		int totalX = 0;
		int totalY = 0;
		int totalZ = 0;

		//������PG���������ͬһ������
		for (int PG = 0; PG < 27; PG++)
		{
			if (X_new[PG].size())
			{

				memcpy(h_cur_X_new, &X_new[PG][0], sizeof(float)*X_new[PG].size());
				memcpy(h_cur_Y_new, &Y_new[PG][0], sizeof(float)*Y_new[PG].size());
				memcpy(h_cur_Z_new, &Z_new[PG][0], sizeof(float)*Z_new[PG].size());
				h_cur_X_new += X_new[PG].size();
				h_cur_Y_new += Y_new[PG].size();
				h_cur_Z_new += Z_new[PG].size();
				totalX += X_new[PG].size();
				totalY += Y_new[PG].size();
				totalZ += Z_new[PG].size();
			}
		}

		hipMemcpy(d_X_new, h_X_new, sizeof(float) * totalX, hipMemcpyHostToDevice);
		hipMemcpy(d_Y_new, h_Y_new, sizeof(float) * totalY, hipMemcpyHostToDevice);
		hipMemcpy(d_Z_new, h_Z_new, sizeof(float) * totalZ, hipMemcpyHostToDevice);

		thrust::counting_iterator<unsigned int> index_sequence_begin(10007);
		int seedValueThreshold = 15;

		for (int K = 0; K < 1; K++)
		{

			thrust::transform(thrust::device, index_sequence_begin, index_sequence_begin + (lenXYZ + 2), d_Random, psrngen(0.0f, 1.0f));
			//hiprandGenerateUniform(gen, d_Random, lenXYZ + 2);
			d_cur_Y_new = d_Y_new;
			d_cur_X_new = d_X_new;
			d_cur_Z_new = d_Z_new;

			for (int PG = 0; PG < 27; PG++)
			{
				if (X_new[PG].size()) {

					dim3 grid((X_new[PG].size() + 1024 - 1) / 1024, 1, 1);
					dim3 block(1024, 1, 1);

					//std::cerr << "Level: " << level << " PG: " << PG << " grid: " << grid.x << " X_New[PG].size " << X_new[PG].size() << std::endl;

					samplingKernel2 << <grid, block >> > (d_cur_X_new, d_cur_Y_new, d_cur_Z_new,
						d_X_new_out, d_Y_new_out, d_Z_new_out,
						d_Random,
						X_new[PG].size(), 1.0 / pow(2, level),
						PG, level,
						d_offset_x, d_offset_y, d_offset_z,
					  d_imagePtr, d_visited, d_compress, width, height, slice, seedValueThreshold);
					//__syncthreads();
					//ÿ��PG�ĵ���ú󣬴���host
					errorCheck = hipGetLastError();
					if (errorCheck != hipSuccess) {
						std::cerr << "Error During Sample" << hipGetErrorString(errorCheck) << std::endl;
						system("pause");
						return -1;
					}
				}
				//ָ���ƶ����¶��ڴ��׵�ַ
				d_cur_X_new += X_new[PG].size();
				d_cur_Y_new += Y_new[PG].size();
				d_cur_Z_new += Z_new[PG].size();
			}
		}

		hipMemcpy(&Xsampled[level][0], d_X_new_out, sizeof(float)*lenXYZ, hipMemcpyDeviceToHost);
		hipMemcpy(&Ysampled[level][0], d_Y_new_out, sizeof(float)*lenXYZ, hipMemcpyDeviceToHost);
		hipMemcpy(&Zsampled[level][0], d_Z_new_out, sizeof(float)*lenXYZ, hipMemcpyDeviceToHost);

		//free�������ڴ�
		std::cerr << "Level " << level << " GPU sampling took me " << timer.getTimerMilliSec() << " milliseconds." << std::endl;
		timer.update();


	}


	std::cerr << "GPU sampling took me " << timer.getTimerMilliSec() << " milliseconds." << std::endl;
	timer.update();

	//�������ɸѡ
	//Filter the samples

	int lastlevel = Xsampled.size() - 1;
	int sampleNum = Xsampled[lastlevel].size();


	filterPoissonSample_gpu(seedArr, center, centerRadius, sampleNum,
		width, height, slice, d_imagePtr, d_X_new_out, d_Y_new_out, d_Z_new_out);

	std::cerr << "GPU filtering took me " << timer.getTimerMilliSec() << " milliseconds." << std::endl;
	timer.update();


	hipFree(d_X_new);
	hipFree(d_Y_new);
	hipFree(d_Z_new);
	hipFree(d_X_new_out);
	hipFree(d_Y_new_out);
	hipFree(d_Z_new_out);
	hipFree(d_Random);
	hipFree(d_offset_x);
	hipFree(d_offset_y);
	hipFree(d_offset_z);
	hipFree(d_visited);


	free(offset_x);
	free(offset_y);
	free(offset_z);
	free(h_X_new);
	free(h_Y_new);
	free(h_Z_new);

	std::cerr << "Free took me " << timer.getTimerMilliSec() << " milliseconds." << std::endl;
	timer.update();


	return 0;
}
