#include "hip/hip_runtime.h"
#include "histogram.h"



__global__
void setZero(int* valueCount, int* valueCountCumulate, int* valueMap)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx >= 256) return;
	valueCount[idx] = 0;
	valueCountCumulate[idx] = 0;
	valueMap[idx] = 0;
}

__global__
void valueCountCal(int *valueCount, uchar *outputPtr,
	int iStart, int jStart, int kStart, int blockSize,
	int height, int width, int slice)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x + iStart;//i ���� iStart ����λ
	int j = threadIdx.y + blockIdx.y * blockDim.y + jStart;
	int k = threadIdx.z + blockIdx.z * blockDim.z + kStart;
	if ((i) >= height || (i - iStart) >= blockSize) return;
	if ((j) >= width || (j - jStart) >= blockSize) return;
	if ((k) >= slice || (k - kStart) >= blockSize) return;

	atomicAdd(&(valueCount[(int)(outputPtr[k * width * height + i * width + j])]), 1);
}

__global__
void valueMapCal(int *valueMap, int blockPixelCount, int *valueCountCumulate)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	valueMap[idx] = (int)((valueCountCumulate[idx] - valueCountCumulate[0]) * 1.0 / (blockPixelCount - valueCountCumulate[0]) * 255);
}

__global__
void blockPixelCountCal(int *valueCountCumulate, int* valueMap, int blockPixelCount)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (valueCountCumulate[idx] < blockPixelCount* 0.99) //ֵ̫С�� ֱ�ӹ���
		valueMap[idx] = 1;
}

__global__
void outputCal(int *valueMap, uchar *outputPtr, uchar *d_block,
	int iStart, int jStart, int kStart, int blockSize,
	int height, int width, int slice)
{
	int j = threadIdx.x + blockIdx.x * blockDim.x + jStart;//i ���� iStart ����λ
	int i = threadIdx.y + blockIdx.y * blockDim.y + iStart;
	int k = threadIdx.z + blockIdx.z * blockDim.z + kStart;
	if (i >= height || (i - iStart) >= blockSize) return;
	if (j >= width || (j - jStart) >= blockSize) return;
	if (k >= slice || (k - kStart) >= blockSize) return;

	uchar temp = outputPtr[k * width * height + i * width + j];
	outputPtr[k * width * height + i * width + j] = valueMap[temp];
}

__global__
void valueCountToCumulateAndvalueMapToZero(int* valueCountCumulate, int* valueCount, int* valueMap)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx == 0)
	{
		valueCountCumulate[idx] = valueCount[idx];
		valueMap[idx] = 0;
	}
}

__global__
void MemsetTo1(int *d_value)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	d_value[idx] = 1;
}

__global__
void blockCal(int *valueMap, uchar *outputPtr, uchar *d_block,//�㵽cpu��
	int iStart, int jStart, int kStart, int blockSize,
	int height, int width, int slice)
{
	int j = threadIdx.x + blockIdx.x * blockDim.x + jStart;//i ���� iStart ����λ
	int i = threadIdx.y + blockIdx.y * blockDim.y + iStart;
	int k = threadIdx.z + blockIdx.z * blockDim.z + kStart;
	if (i >= height || (i - iStart) >= blockSize) return;
	if (j >= width || (j - jStart) >= blockSize) return;
	if (k >= slice || (k - kStart) >= blockSize) return;

	uchar temp = outputPtr[k * width * height + i * width + j];

	int blockHeight = MIN(blockSize, height - iStart);
	int blockWidth = MIN(blockSize, width - jStart);

	d_block[(j - jStart) + (i - iStart) * blockWidth + (k - kStart) * blockHeight * blockWidth] = temp;//��output���һ��ͼ���е�������һ���ڴ� block ���棬 �ٰ�����֮���output�滻 �����Ϳ��Եõ���Ҫ��ͼ����
}

__global__
void histogramComplete(uchar *d_C, int *d_D, int *valueCountCumulate)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= 256) return;
	if (!d_D[idx]) return;
	int index = d_C[idx];
	valueCountCumulate[index] = d_D[idx];
}

void equalizeHistogramGpu(uchar* const inputPtr, uchar* const outputPtr, int width, int height, int slice, int *sizes, uchar* d_output)
{

	int *valueCount, *valueCountCumulate, *valueMap;
	int blockSize = 256;
	int blockPixelCount = 0;
	//const int blockNum = (width / blockSize + 1) * (height / blockSize + 1) * (slice / blockSize + 1);
	//Modified by jifaley 20210830
	const int blockNum = ((width - 1) / blockSize + 1) * ((height - 1) / blockSize + 1) * ((slice - 1) / blockSize + 1);
	std::cout << "blockNum: " << blockNum << std::endl;
	hipMalloc((void**)&valueCount, sizeof(int) * 256 * blockNum);
	hipMalloc((void**)&valueCountCumulate, sizeof(int) * 256 * blockNum);
	hipMalloc((void**)&valueMap, sizeof(int) * 256 * blockNum);

	//hipHostMalloc((void**)&d_output, sizeof(uchar)*(width*height*slice));//pine

	//hipMemset(d_value, 0,sizeof(int)*(width*height*slice));

	hipMemcpy(d_output, inputPtr, sizeof(uchar)*(width*height*slice), hipMemcpyHostToDevice);
	//std::string save_dir_f_ = "image(single)/neuron01_kernel_before.tif";
	//saveTiff(save_dir_f_.c_str(), inputPtr, &sizes[0]);

	//d_block:������ d_C d_D ���� reduece_by_key
	uchar *d_block;
	int *d_value;
	uchar *d_C;
	int *d_D;

	hipMalloc((void**)&d_block, sizeof(uchar)*(width*height*slice));
	hipMalloc((void**)&d_value, sizeof(int)*blockSize*blockSize*blockSize);
	hipMalloc((void**)&d_C, sizeof(uchar)*blockSize*blockSize*blockSize);
	hipMalloc((void**)&d_D, sizeof(int)*blockSize*blockSize*blockSize);
	//set some streams to overlap the kernels or the memcpy
	//����ֻ����512 512 512 ͼƬ�� �����ѹ̶��ߴ� 128
	//const int MAX_NUM_STREAMS = 1480;
	//hipStream_t streams[MAX_NUM_STREAMS];
	//for (int n = 0; n < MAX_NUM_STREAMS; ++n)
	//{
	//	hipStreamCreate(&(streams[n]));
	//}
	//int STREAMS = 0;
	//����d_block������ offset

	//Modified by jifaley 20210830
	int offset = 0;
	//value_offset
	int value_offset = 0;

	//�ĳɶ�� for loop ��ָ��stream
//#pragma unroll

	thrust::device_ptr<uchar> d_block_thrust(d_block);
	thrust::device_ptr<int> d_value_thrust(d_value);

	for (int kStart = 0; kStart < slice; kStart += blockSize)
		for (int iStart = 0; iStart < height; iStart += blockSize)
			for (int jStart = 0; jStart < width; jStart += blockSize) {
				//offset += blockSize * blockSize*blockSize;


				//01
				dim3 block_size(256, 1, 1);
				dim3 grid_size(1, 1, 1);
				setZero << <grid_size, block_size >> > (valueCount + value_offset, valueCountCumulate + value_offset, valueMap + value_offset);

				//02
				dim3 grid_size_blockCal(1, MIN(blockSize, height - iStart), MIN(blockSize, width - jStart));
				//��
				dim3 block_size_blockCal(MIN(blockSize, slice - kStart), 1, 1);

				blockCal << <grid_size_blockCal, block_size_blockCal >> > (valueMap + value_offset, d_output, d_block + offset,
					iStart, jStart, kStart, blockSize,
					height, width, slice);

				//03
				blockPixelCount = MIN(blockSize, slice - kStart) * MIN(blockSize, height - iStart) * MIN(blockSize, width - jStart);

				dim3 block_size_1(256, 1, 1);
				dim3 grid_size_1((blockPixelCount + 256 - 1) / 256, 1, 1);

				//����ֱ��ͼʱ�õ� sort_by_key ���� key ȫΪһ
				MemsetTo1 << <grid_size_1, block_size_1 >> > (d_value);

				//04
				blockPixelCount = MIN(blockSize, slice - kStart) * MIN(blockSize, height - iStart) * MIN(blockSize, width - jStart);
				thrust::sort(thrust::device, d_block + offset, d_block + offset + blockPixelCount);


				//05
				blockPixelCount = MIN(blockSize, slice - kStart) * MIN(blockSize, height - iStart) * MIN(blockSize, width - jStart);

				thrust::equal_to<uchar> binary_pred;
				thrust::reduce_by_key(thrust::device, d_block + offset, d_block + offset + blockPixelCount, d_value, d_C, d_D, binary_pred);

				//��d_C d_D ӳ�������������
				//uchar *d_blockHistogram;  ԭ�������¿�һ��  d_blockHistogram  ����ֱ�������ϵ� valueCountCumulate
				//hipMalloc((void**)&d_blockHistogram, sizeof(uchar) * 256);
				dim3 blockComplete(256, 1, 1);
				dim3 gridComplete(1, 1, 1);
				histogramComplete << <gridComplete, blockComplete >> > (d_C, d_D, valueCountCumulate + value_offset);

				thrust::inclusive_scan(thrust::device, valueCountCumulate + value_offset, valueCountCumulate + value_offset + 256, valueCountCumulate + value_offset);

				//06
				dim3 block_size_3(256, 1, 1);
				dim3 grid_size_3(1, 1, 1);
				valueMapCal << <grid_size_3, block_size_3 >> > (valueMap + value_offset, blockPixelCount, valueCountCumulate + value_offset);

				//07
				dim3 block_size_4(256, 1, 1);
				dim3 grid_size_4(1, 1, 1);
				blockPixelCountCal << <grid_size_4, block_size_4 >> > (valueCountCumulate + value_offset, valueMap + value_offset, blockPixelCount);


				//08
				dim3 grid_size_5(blockSize / 16, blockSize / 16, blockSize / 4);
				dim3 block_size_5(16, 16, 4);

				outputCal << <grid_size_5, block_size_5 >> > (valueMap + value_offset, d_output, d_block + offset,
					iStart, jStart, kStart, blockSize,
					height, width, slice);

				//Modified by jifaley 20210830
				offset += MIN(blockSize, height - iStart) * MIN(blockSize, width - jStart)*MIN(blockSize, slice - kStart);
				value_offset += 256;
			}


	hipFree(d_block);
	hipFree(d_value);
	hipFree(d_C);
	hipFree(d_D);
	hipMemcpy(outputPtr, d_output, sizeof(uchar)*(width*height*slice), hipMemcpyDeviceToHost);
	//int outputPtrInt = (int)outputPtr[6558462];
	//printf("outputPtr[6558462] = %d\n", outputPtrInt);
	//std::string save_dir_f = "image(single)/neuron01_kernel_after.tif";
	//saveTiff(save_dir_f.c_str(), outputPtr, &sizes[0]);

	hipFree(valueCount);
	hipFree(valueCountCumulate);
	hipFree(valueMap);
	//Modified by jifaley 20210830
	//free(inputPtr); �����ͷţ����滹Ҫ��
}
