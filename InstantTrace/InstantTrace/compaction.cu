#include "hip/hip_runtime.h"
#include "compaction.h"
#include "TimerClock.hpp"

template<typename T>
struct is_non_zero {
	__host__ __device__
		bool operator()(T x) const
	{
		return x != 0;
	}
};

template<typename T>
struct is_zero {
	__host__ __device__
		bool operator()(T x) const
	{
		return x == 0;
	}
};


using thrust::placeholders::_1;

//getCompressMap:compactImage���Ӻ��������ڼ�����ѹ�����ӳ�䡣d_compressΪԪ���±�->ѹ���±꣬d_decompress��֮��
//getCompressMap:The sub-function of compactImage. Calculating the mapping for stream compaction. The "d_compress" array
//is the mapping from the original element index to the compressed element index. The "d_decompress" array is the inversed mapping.
__global__
void getCompressMap(int* d_compress, int* d_decompress, uchar* d_imagePtr, uchar* d_imagePtr_compact, int newSize)
{
	int smallIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (smallIdx >= newSize) return;
	int fullIdx = d_decompress[smallIdx];

	d_compress[fullIdx] = smallIdx;
	d_imagePtr_compact[smallIdx] = d_imagePtr[fullIdx];
}

/*
������compactImage
���ܣ�ѹ��ԭͼ��ȥ����0���֡� 
�����d_compactedImagePtr(ѹ�����ͼ)��d_compress (ԭͼ->ѹ��ͼӳ��)��d_decompress(ѹ��ͼ->ԭͼӳ�䣩
˼·�����Ƚ��������غ����±��Ϊtuple��������(0,value0), (1, value1), (2,value2)....
������value< 0�Ĳ���ɾ����ʣ���tuple��Ϊ: (id0, value_id0), (id1, value_id1)...
��ô,ʣ���valueֵ��Ϊѹ�����ͼ��ʣ���id��Ϊѹ�����ֵ��Ӧ��ԭͼ�е��±ꡣ
ʵ�֣�ʹ��thrust���copy_if ���� remove_if ����
*/
/*
Function��compactImage
Work��Compress the original image, leave out the zero-valued elements. (Also known as Stream Compaction)
Output��d_compactedImagePtr(The compressed image)��d_compress (The compression mapping)��d_decompress(The decompression mapping)
Implementaion��Binding the voxels and their indices to tuples, as the form of (0,value0), (1, value1), (2,value2)....
After deleting the zero-valued tuples, the remainders are arranged as (id0, value_id0), (id1, value_id1)...
Thus, these values form the compressed image, and these ids are the corresponding indices in the orginal image.
This function can be implemented by thrust::copy_if or thrust::remove_if.
*/

void compactImage(uchar* d_imagePtr, uchar* &d_imagePtr_compact, int* &d_compress, int* &d_decompress, int width, int height, int slice, int& newSize)
{
	TimerClock timer;
	timer.update();

	hipError_t errorCheck;
	hipMalloc(&d_compress, sizeof(int) * width * height * slice);
	int* d_sequence = d_compress; //ԭ�����������顣Ϊ�˽�ʡ��������ʱ����ͬһ��ռ�

	//������50ms���ҵ�ͬ��ʱ�䣨��ʹȥ��cuDeiveSyncronize()��hipMemset()Ҳ��ǿ��ͬ����
	hipDeviceSynchronize();
	std::cerr << "stream compaction preprocess cost: " << timer.getTimerMilliSec() << "ms" << std::endl;
	timer.update();


	//����copy_if��d_sequence�����µ���ԭʼ�����ݷ�0ֵ���±ꡣ�ò�����stable�ġ� newSize��Ϊ��0ֵ�ĸ�����
	try
	{
		int* d_copy_end = thrust::copy_if(thrust::device, thrust::make_counting_iterator<int>(0), thrust::make_counting_iterator<int>(width * height * slice), d_imagePtr, d_sequence, _1 != 0);
		newSize = d_copy_end - d_sequence;
	}
	catch (thrust::system_error error)
	{
		std::cerr << std::string(error.what()) << std::endl;
	}

	hipMalloc(&d_decompress, sizeof(int) * newSize);
	hipMalloc(&d_imagePtr_compact, sizeof(uchar) * newSize);
	hipMemcpy(d_decompress, d_sequence, sizeof(int) * newSize, hipMemcpyDeviceToDevice);
	hipMemset(d_compress, 0xff, sizeof(int) * width * height * slice);

	//�����Ӧ��ӳ��
	getCompressMap << < (newSize - 1) / 256 + 1, 256 >> > (d_compress, d_decompress, d_imagePtr, d_imagePtr_compact, newSize);

	errorCheck = hipGetLastError();
	if (errorCheck != hipSuccess) {
		std::cerr << "Duing copyif " << hipGetErrorString(errorCheck) << std::endl;
		system("pause");
		return;
	}
	//�������㣬����copy_if ��getMap()��ʵ�ʺ�ʱԼ20ms����������50ms��ͬ���������ۡ�
}

struct isValid_functor {

	const uchar threshold;

	isValid_functor(uchar _th) : threshold(_th) {}
	__host__ __device__
		bool operator()(const uchar& x) const
	{
		return x >= threshold;
	}
};


//���ܣ�����d_sequence������Ԫ�ص�x,y,z����ƽ��ֵ��Ȼ���ҵ�����ƽ��ֵ�����Ԫ�ء�
//Work��Calculating the average of x,y,z coordinates in the d_sequence array��and find the element nearest to this average coordinate.
__global__
void centerProcess(int* d_sequence, int* d_decompress, int maxSeedNum, int width, int height, int slice)
{
	__shared__ int sumX, sumY, sumZ;
	__shared__ int minDist;
	__shared__ int minPos;


	int tid = threadIdx.x;
	if (tid >= maxSeedNum) return;

	int smallIdx = d_sequence[tid];
	int fullIdx = d_decompress[smallIdx];

	int z = fullIdx / (width * height);
	int y = fullIdx % (width * height) / width;
	int x = fullIdx % width;

	atomicAdd(&sumZ, z);
	atomicAdd(&sumY, y);
	atomicAdd(&sumX, x);

	__syncthreads();

	if (tid == 0)
	{
		sumX = sumX / maxSeedNum;
		sumY = sumY / maxSeedNum;
		sumZ = sumZ / maxSeedNum;
		minDist = 2147483647;
		d_sequence[0] = 2147483647;
	}

	__syncthreads();

	int dist = sqrtf((sumZ - z) * (sumZ - z) + (sumY - y) * (sumY - y) + (sumX - x) * (sumX - x));

	atomicMin(&minDist, dist);

	__syncthreads();

	if (minDist == dist)
	{
		atomicMin(&d_sequence[0], fullIdx);
	}
}

/*
������getCenterPos
���ܣ�Ѱ��Radius���ĵ㣬��Ϊ����(soma)
�����maxPos(�����λ��)��maxRadius(���뾶)
˼·�����ֻ�Ұ뾶���ĵ㣬���ܻ��кܶ���ͬ��ȡֵ������ƫб��
��ˣ����ǽ���Χ�뾶�㹻������ɵ��λ�ü���ƽ��ֵ����Ϊ�µİ������ġ�
ʵ�֣�ʹ��thrust���copy_if ���� remove_if ����
*/
/*
Function��getCenterPos
Work��Find the point with the largest radius, as the center of neuron soma.
Output��maxPos(the location of soma)��maxRadius(the largest radius)
Implementation��The element with the largest radius may not locates at the neuron center.
We generate a lot of candidates with large radius, and calculate the center of them as the neuron center.
*/
void getCenterPos(int* d_compress, int* d_decompress, uchar* d_radiusMat_compact, int width, int height, int slice, int newSize, int&maxPos, int& maxRadius)
{
	thrust::device_ptr<uchar> d_ptr(d_radiusMat_compact);
	thrust::device_ptr<uchar> iter = thrust::max_element(d_ptr, d_ptr + newSize);
	maxRadius = *iter;
	//����ͨ��max_element��������뾶��ֵ
	//Find the largest radius

	printf("Init maxRadius: %d\n", maxRadius);
	
	int* d_sequence;
	hipMalloc(&d_sequence, sizeof(int) * newSize);

	//���ǽ����뾶��4/5�������뾶-5��Ϊ��ֵ��ѡ��һЩ��ѡ�㣻����Щ��ѡ���������Ϊ�������ġ�
	//The threshold radius for generating center candidates
	uchar thresholdRadius = MAX(maxRadius * 4 / 5, maxRadius - 5);

	int* d_copy_end = thrust::copy_if(thrust::device, thrust::make_counting_iterator<int>(0), thrust::make_counting_iterator<int>(newSize), d_radiusMat_compact, d_sequence,isValid_functor(thresholdRadius));
	int maxSeedNum = d_copy_end - d_sequence;

	maxSeedNum = MIN(maxSeedNum, 512);

	//����d_sequence������Ԫ�ص�x,y,z����ƽ��ֵ��Ȼ���ҵ�����ƽ��ֵ�����Ԫ�ء�
	centerProcess << <1, maxSeedNum >> > (d_sequence, d_decompress, maxSeedNum, width, height, slice);

	thrust::device_ptr<int> dp(d_sequence);

	maxPos = *dp;
	hipFree(d_sequence);
}