#include "hip/hip_runtime.h"
#include "mergeSegments.h"
#include "fastmarching.h"
#include "TimerClock.hpp"

////If one want to test the result without merging, use this option.
//#define __NO__MERGE

//disjoint set
//���鼯
__device__ int getfather_gpu(int* d_disjointSet, int x)
{
	if (d_disjointSet[x] == x) return x;
	return d_disjointSet[x] = getfather_gpu(d_disjointSet, d_disjointSet[x]);
}

__device__ void merge_gpu(int* d_disjointSet, uchar* d_seedRadiusMat, int x, int y)
{
	int fa_x = getfather_gpu(d_disjointSet, x);
	int fa_y = getfather_gpu(d_disjointSet, y);
	//�뾶���Ϊ���ף�
	//��ͬ��������С��Ϊ����

	int rx = d_seedRadiusMat[fa_x];
	int ry = d_seedRadiusMat[fa_y];

	if (rx > ry)
	{
		d_disjointSet[fa_y] = fa_x;
	}
	else if (rx < ry)
	{
		d_disjointSet[fa_x] = fa_y;
	}
	else if (rx == ry)
	{
		if (fa_x < fa_y)
			d_disjointSet[fa_y] = d_disjointSet[x];
		else
			d_disjointSet[fa_x] = d_disjointSet[y];
	}
}


/*
����:findInterSectKernel
����:����鿴ĳ�����Ƿ���������֧�Ľ���㣬���ҽ����еĽ�������һ�������С�
�ж��Ƿ��ǽ���㣺����õ��top-2��Ӧ�ķ�֧��top-1��Ӧ�ķ�֧��ͬ(�������fastmarching����)��
���ڶ�����Ҫ����ԭ�Ӳ���,���ѡ����ʹ��share memory����ÿ��block�ڲ�����һ��С�Ͷ��У�����ٺϲ��������С�
mergeSegments()���в���������<50ms�����û�н���������Ϊ��ѹ����
d_seedNumberPtr: ��¼�ýڵ����ĸ�������չ����
*/
/*
Funciton:findInterSectKernel
Work:Checking if a node is an intersect of two neuron branches. All of the intersects are stored into an array(or,queue).
Implementation: If the node's top-2 parent and top-1 parent are extended from different seeds (see fastmarching.cu for detail)
it is regarded as an intersect.
The queue operations are based on atomic operations. We optimized this implementation using atomic operations in shared memory:
in each block, we build a small queue using shared-memory-atomic operations, and combine these small blocks to a global array.
These atomic operations can be replaced by stream compaction, but the whole mergeSegments() function runs very fast, so there 
is little performance gain can be reach by stream compaction.
d_seedNumberPtr: The current voxel is extended from which seed.
*/
__global__ void findInterSectKernel(int * d_compress, int* d_decompress, int* d_parentPtr_compact, short int* d_seedNumberPtr, int width, int height, int slice, int newSize, int* queue, int* queueHead, int* queueLock, int queueMaxSize)
{
	int smallIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (smallIdx >= newSize) return;
	__shared__ int localQueue[512];
	__shared__ int localQueueHead[1];
	__shared__ int localQueueLock[1];
	__shared__ int offset[1];

	if (threadIdx.x == 0)
	{
		*localQueueHead = 0;
		*localQueueLock = 0;
	}
	__syncthreads();

	int queueloop;
	int parent2SmallIdx = d_parentPtr_compact[smallIdx + newSize];

	if (parent2SmallIdx != -1)
	{
		int ra = d_seedNumberPtr[smallIdx];
		int rb = d_seedNumberPtr[parent2SmallIdx];

		if (ra != 0 && rb != 0 && ra != rb)
		{
			queueloop = 0;
			do {
				if (queueloop = atomicCAS(localQueueLock, 0, 1) == 0)
				{
					int localQsize = localQueueHead[0];
					localQueue[localQsize] = smallIdx;
					localQsize += 1;
					localQueueHead[0] = localQsize;
				}
				__threadfence_block();
				if (queueloop) atomicExch(localQueueLock, 0);
			} while (!queueloop);
		}
	}


	__syncthreads();

	//���������н���ͳ�ƣ���ÿ��С��ֱ����offset
	int localNum = *localQueueHead;
	if (localNum == 0) return;

	if (threadIdx.x == 0 && localNum != 0)
	{
		queueloop = 0;
		do {
			if (queueloop = atomicCAS(queueLock, 0, 1) == 0)
			{
				int qSize = *queueHead;

				if (qSize + localNum < queueMaxSize)
				{
					*offset = qSize;
					qSize += localNum;
					*queueHead = qSize;
				}
				else
				{
					//�Ų����ˣ�������
					*localQueueHead = 0;
				}
			}
			__threadfence();
			if (queueloop) atomicExch(queueLock, 0);
		} while (!queueloop);
	}
	__syncthreads();

	//��share memory����Ķ����������ܶ�����
	//combine the local queues to a global array
	if (threadIdx.x < *localQueueHead)
	{
		queue[*offset + threadIdx.x] = localQueue[threadIdx.x];
	}
}

/*
����:chlcChldKernel
����:�������ÿ�����child����(��������)
*/
//renew the number of child of the nodes
__global__ void calcChildKernel(int* d_compress, int* d_decompress, int* d_parentMat, int* d_childNumPtr, int width, int height, int slice, int newSize)
{
	int smallIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (smallIdx >= newSize) return;

	int parentSmallIdx = d_parentMat[smallIdx];
	if (parentSmallIdx == -1 || parentSmallIdx == smallIdx) return;
	atomicAdd(d_childNumPtr + parentSmallIdx, 1);
}

/*
����:interSectCheckKernel
����:����鿴ÿ��������Ƿ���Ч��
��Ч�ı�׼�����Top-1 parent�� Top-2 parent ���Ÿ�����Ϣһ·�����Ƿֱ��Ӧ�ĸ�ǰ����
���һ·�����ڵķ�֧��û�з����ı䣬������Ч�ġ������ڽ������ϲ�������֧ʱ������ǣ����������֧�Ӷ���������
*/
/*
Function:interSectCheckKernel
Work:Checking if one intersect is valid.
Implementation��The checking are started from the top-1 and top-2 parent of the intersect. Moving along the chind->parent 
edge until reach the seed point. If all of the voxels along this path belongs to the same seed, the node is a valid intersect.
Otherwise, errors may occur when merging the two branches.
*/
__global__ void interSectCheckKernel(int* d_compress, int* d_decompress, int* d_interSectArr, int interSectNum, uchar* d_interSectValid, int* d_parentMat, short int* d_seedNumberPtr, int* counter, int width, int height, int slice, int newSize)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= interSectNum) return;
	int smallIdx = d_interSectArr[idx];
	
	int parentSmallIdx = d_parentMat[smallIdx];
	int parent2SmallIdx = d_parentMat[smallIdx + newSize];


	int curSeed = d_seedNumberPtr[smallIdx];
	int parentSeed = d_seedNumberPtr[parentSmallIdx];
	int parent2Seed = d_seedNumberPtr[parent2SmallIdx];

	bool checkIfValidInterSect = true;
	int curSmallIdxTemp = smallIdx;
	while(d_parentMat[curSmallIdxTemp] != curSmallIdxTemp)
	{
		int curSeedTemp = d_seedNumberPtr[curSmallIdxTemp];
		if (curSeedTemp != curSeed)
		{
			checkIfValidInterSect = false;
			break;
		}

		curSmallIdxTemp = d_parentMat[curSmallIdxTemp];
	}

	curSmallIdxTemp = parent2SmallIdx;
	while (d_parentMat[curSmallIdxTemp] != curSmallIdxTemp)
	{
		int curSeedTemp = d_seedNumberPtr[curSmallIdxTemp];
		if (curSeedTemp != parent2Seed)
		{
			checkIfValidInterSect = false;
			break;
		}
		curSmallIdxTemp = d_parentMat[curSmallIdxTemp];
	}
	if (checkIfValidInterSect)
	{
		d_interSectValid[idx] = 1;
		atomicAdd(counter, 1);
	}
	else
	{
		d_interSectValid[idx] = 0;
	}

}

/*
����:interSectProcessKernel
����:���ϲ�������֧ʱ����һ����֧��father��Ϣ���¡�
*/
/*
Function:interSectProcessKernel
Work:Renew the "father"(or parent) information of the branches when merging.
*/
__global__ void interSectProcessKernel(int* d_compress, int* d_decompress, int* d_interSectArr, int interSectNum, uchar* d_interSectValid, int* d_parentMat, short int* d_seedNumberPtr, uchar* d_seedRadiusMat, int* d_disjointSet, int width, int height, int slice, int newSize)
{
	if (threadIdx.x != 0) return;
	for (int it = 0; it < interSectNum; it++)
	{
		if (d_interSectValid[it] == 0)
			continue;
		
		int curSmallIdx = d_interSectArr[it];
		int parentSmallIdx = d_parentMat[curSmallIdx];
		int parent2SmallIdx = d_parentMat[curSmallIdx + newSize];

		int curSeed = d_seedNumberPtr[curSmallIdx];
		int parentSeed = d_seedNumberPtr[parentSmallIdx];
		int parent2Seed = d_seedNumberPtr[parent2SmallIdx];

		int father1 = getfather_gpu(d_disjointSet, curSeed);
		int father2 = getfather_gpu(d_disjointSet, parent2Seed);
		int prevIdxTemp, nextIdxTemp, curIdxTemp;
		//printf("%d %d %d %d %d %d\n", it, curIdx, father1, father2, d_radiusMat[father1], d_radiusMat[father2]);
		//father��һ����Ҫ�ϲ�
		//merge branches have different father in the disjoint set
		if (father1 != father2)
		{
			int r1 = d_seedRadiusMat[father1];
			int r2 = d_seedRadiusMat[father2];
			//std::cerr << "Merge:" << father1 << ' ' << father2 << std::endl;
			//�뾶��������С�ĵ���,��parent2��ʼ,�𽥷�����,parnet2��parent��Ϊcur
			//The branch with larger seed radius or smaller seed index becomes the new root of the merged branch.

			//The merging 
			//if (father1 < father2)
			if (r1 > r2 || (r1 == r2 && father1 < father2))
			{

				curIdxTemp = parent2SmallIdx;
				prevIdxTemp = curSmallIdx;
				while (d_parentMat[curIdxTemp] != curIdxTemp) //|| parentSeedMat[curIdxTemp] != father2)
				{
					nextIdxTemp = d_parentMat[curIdxTemp];

					//1.�޸�
					//1.modify
					d_parentMat[curIdxTemp] = prevIdxTemp;
					//2.ǰ��
					//2.forward
					prevIdxTemp = curIdxTemp;
					curIdxTemp = nextIdxTemp;
				}
				//ȷ���ߵ�root��root�����parent�յ����޸�
				//Move along the path until the seed point is reached
				if (d_parentMat[curIdxTemp] == curIdxTemp && curIdxTemp != prevIdxTemp)
				{
					d_parentMat[curIdxTemp] = prevIdxTemp;
				}
			}
			//else if (father1 > father2)
			else
			{
				curIdxTemp = curSmallIdx;
				prevIdxTemp = parent2SmallIdx;

				while (d_parentMat[curIdxTemp] != curIdxTemp) //|| parentSeedMat[curIdxTemp] != father2)
				{
					nextIdxTemp = d_parentMat[curIdxTemp];

					//1.�޸�
					d_parentMat[curIdxTemp] = prevIdxTemp;
					//2.ǰ��
					prevIdxTemp = curIdxTemp;
					curIdxTemp = nextIdxTemp;
				}
				//ȷ���ߵ�root��root�����parent�յ����޸�
				if (d_parentMat[curIdxTemp] == curIdxTemp && curIdxTemp != prevIdxTemp)
				{
					d_parentMat[curIdxTemp] = prevIdxTemp;
				}
			}

			merge_gpu(d_disjointSet, d_seedRadiusMat, father1, father2);
		}
	}
}

//Ϊ�˷�ֹ���鼯�Ľ�������£������е��ٴν���getfather()
//Renew the informatiion in the disjoint set
__global__ void renewColorKernel(int totalColor, int* d_disjointSet)
{
	if (threadIdx.x != 0) return;
	for (int i = 0; i < totalColor; i++)
	{
		getfather_gpu(d_disjointSet, i);
	}
}

__global__
void getSeedRadius(int* d_seedArr, int* d_compress, uchar* d_seedRadiusMat, uchar* d_radiusMat_compact, int totalColor)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= totalColor) return;
	int fullIdx = d_seedArr[idx];
	int smallIdx = d_compress[fullIdx];
	d_seedRadiusMat[idx] = d_radiusMat_compact[smallIdx];
}


__global__
void getExtraIntersect(int* d_compress, int* d_decompress, int* d_childNumMat, int* d_parentPtr_compact, short int* d_seedNumberPtr, int width, int height, int slice, int newSize)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= newSize) return;
	int smallIdx = idx;

	if (d_childNumMat[smallIdx] > 0) return;
	if (d_parentPtr_compact[smallIdx + newSize] != -1) return;
	int fullIdx = d_decompress[smallIdx];



	int3 curPos;
	curPos.z = fullIdx / (width * height);
	curPos.y = fullIdx % (width * height) / width;
	curPos.x = fullIdx % width;

	int3 neighborPos;
	int neighborIdx;
	int neighborSmallIdx;
	uchar neighborValue;

	int curSeed = d_seedNumberPtr[smallIdx];

	//����Χ26��������չ
	//Extend towards 26-way neighbor
	for (int k = 0; k < 6; k++)
	{
		neighborPos.x = curPos.x + dx3dconst[k];
		neighborPos.y = curPos.y + dy3dconst[k];
		neighborPos.z = curPos.z + dz3dconst[k];

		if (neighborPos.x < 0 || neighborPos.x >= width || neighborPos.y < 0 || neighborPos.y >= height
			|| neighborPos.z < 0 || neighborPos.z >= slice)
			continue;

		neighborIdx = neighborPos.z * width * height + neighborPos.y * width + neighborPos.x;
		neighborSmallIdx = d_compress[neighborIdx];
		if (neighborSmallIdx == -1)	continue;


		if (d_parentPtr_compact[neighborSmallIdx + newSize] != -1) continue;

		int newSeed = d_seedNumberPtr[neighborSmallIdx];

		if (curSeed != newSeed)
		{
			if (curSeed < newSeed)
			{
				d_parentPtr_compact[smallIdx + newSize] = neighborSmallIdx;
				break;
			}
		}
	}


}

/*
����:mergeSegments
����:�ڳ�ʼ׷�����֮�󣬶��������񾭷�֧���кϲ�
����:���ӵ㼯�� seedArr, ԭͼd_imagePtr, �뾶��Ϣd_radiusMat, 
��֧�������ӵ���Ϣd_seedNumberPtr, ���鼯d_disjointset
*/
/*
Function:mergeSegments
Work:merge the connected branches after constructing initial neuron.
Input: seedArr(The seed set), d_imagePtr(The image), d_radiusMat(The control radius of voxels),
d_seedNumberPtr(The voxel is extended from which seed point), d_disjointset
*/
void mergeSegments(std::vector<int>& seedArr, std::vector<int>& disjointSet, int width, int height, int slice, int newSize, uchar* d_imagePtr, uchar* d_imagePtr_compact, int* d_compress, int* d_decompress, int* d_childNumMat, uchar* d_radiusMat_compact, int* d_parentPtr_compact, short int* d_seedNumberPtr, int* d_disjointSet)
{
	TimerClock timer;
	timer.update();
	std::vector<int> intersectArr;
	hipError_t errorCheck;

	int* d_seedArr;
	hipMalloc(&d_seedArr, sizeof(int) * seedArr.size());
	hipMemcpy(d_seedArr, &(seedArr[0]), sizeof(int) * seedArr.size(), hipMemcpyHostToDevice);
	int seedNum = seedArr.size();



	//û��:4500
	getExtraIntersect << <(newSize - 1) / 256 + 1, 256 >> > (d_compress, d_decompress, d_childNumMat, d_parentPtr_compact, d_seedNumberPtr, width, height, slice, newSize);
	errorCheck = hipGetLastError();
	if (errorCheck != hipSuccess) {
		std::cerr << "After Get Extra Intersect " << hipGetErrorString(errorCheck) << std::endl;
		system("pause");
		return;
	}

	std::cerr << "Get Extra Intersect cost: " << timer.getTimerMilliSec() << "ms" << std::endl;
	timer.update();


	//�ж�һ��ÿ�����㸽���Ƿ���������������

	//01 ����InterSect
	//01 Finding the intersect of branches

	
	const int queueSize = 5000000; //max number of intersects
	int* queue = (int*)malloc(sizeof(int) * queueSize);
	int* d_queue;
	hipMalloc(&d_queue, sizeof(int) * queueSize);
	int* d_queueHead;
	int* d_queueLock;
	hipMalloc(&d_queueHead, sizeof(int));
	hipMalloc(&d_queueLock, sizeof(int));
	hipMemset(d_queueHead, 0, sizeof(int));
	hipMemset(d_queueLock, 0, sizeof(int));	
	findInterSectKernel << <(newSize - 1) / 256 + 1, 256 >> > (d_compress, d_decompress, d_parentPtr_compact, d_seedNumberPtr, width, height, slice, newSize, d_queue, d_queueHead, d_queueLock, queueSize);
	int* qSize = (int*)malloc(sizeof(int));
	hipMemcpy(qSize, d_queueHead, sizeof(int), hipMemcpyDeviceToHost);
	intersectArr.resize(*qSize);
	hipMemcpy(&(intersectArr[0]), d_queue, sizeof(int) * (*qSize), hipMemcpyDeviceToHost);
	int interSectNum = *qSize;

	errorCheck = hipGetLastError();
	if (errorCheck != hipSuccess) {
		std::cerr << "After InterSect Finding " << hipGetErrorString(errorCheck) << std::endl;
		system("pause");
		return;
	}
	std::cerr << "InterSect Size: " << interSectNum << std::endl;
	std::cerr << "InterSect Finding cost: " << timer.getTimerMilliSec() << "ms" << std::endl;
	timer.update();




	//02 ���InterSect
	//02 Checking if the intersect is valid (only relate to two branches)
	int countValidInterSect = 0;
	int * d_parentMat_compact = d_parentPtr_compact;

	int* counter = (int*)malloc(sizeof(int));
	int* d_counter;
	hipMalloc(&d_counter, sizeof(int));
	hipMemset(d_counter, 0, sizeof(int));
	uchar* d_interSectValid;
	hipMalloc(&d_interSectValid, sizeof(uchar) * interSectNum);
	hipMemset(d_interSectValid, 0, sizeof(uchar) * interSectNum);
	interSectCheckKernel << <(interSectNum - 1) / 256 + 1, 256 >> > (d_compress, d_decompress, d_queue, interSectNum, d_interSectValid, d_parentPtr_compact, d_seedNumberPtr, d_counter, width, height, slice, newSize);
	
	
	//hipMemcpy(counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	errorCheck = hipGetLastError();
	if (errorCheck != hipSuccess) {
		std::cerr << "After InterSect Checking " << hipGetErrorString(errorCheck) << std::endl;
		system("pause");
		return;
	}


	//std::cerr << "Valid InterSect: " << *counter << std::endl;
	std::cerr << "InterSect Chekcing cost: " << timer.getTimerMilliSec() << "ms" << std::endl;
	timer.update();

#ifdef __NO__MERGE
	//�������Բ��ϲ��Ľ����ʹ������Ĵ���
	//If one want to test the result without merging, use this option.
	hipMemset(d_interSectValid, 0, sizeof(uchar) * interSectNum);
#endif // __NO__MERGE


	//03 Merge
	int totalColor = seedArr.size(); 
	//Hint: The indices of seeds are started from 1. The 0th seed is a dummy seed.
	uchar* d_seedRadiusMat;
	hipMalloc(&d_seedRadiusMat, sizeof(int) * totalColor);

	getSeedRadius << <(totalColor - 1) / 32 + 1, 32 >> > (d_seedArr, d_compress, d_seedRadiusMat, d_radiusMat_compact, totalColor);


	interSectProcessKernel << <1, 1 >> > (d_compress, d_decompress, d_queue, interSectNum, d_interSectValid, d_parentPtr_compact, d_seedNumberPtr, d_seedRadiusMat, d_disjointSet, width, height, slice, newSize);
	renewColorKernel << <1, 1 >> > (totalColor, d_disjointSet);

	hipMemcpy(&(disjointSet[0]), d_disjointSet, sizeof(int) * totalColor, hipMemcpyDeviceToHost);
	
	hipDeviceSynchronize();
	errorCheck = hipGetLastError();
	if (errorCheck != hipSuccess) {
		std::cerr << "After Merge Reverse: " << hipGetErrorString(errorCheck) << std::endl;
		system("pause");
		return;
	}
	std::cerr << "Merging cost: " << timer.getTimerMilliSec() << "ms" << std::endl;
	timer.update();

	//04 ����ͳ��childNum
	//04 Renew the number of childs

	hipMemset(d_childNumMat, 0, sizeof(int) * newSize);
	calcChildKernel << <(newSize - 1) / 256 + 1, 256 >> > (d_compress, d_decompress, d_parentPtr_compact, d_childNumMat, width, height, slice, newSize);
	
	
	hipDeviceSynchronize();
	errorCheck = hipGetLastError();
	if (errorCheck != hipSuccess) {
		std::cerr << "Renew ChildNum : " << hipGetErrorString(errorCheck) << std::endl;
		system("pause");
		return;
	}
	std::cerr << "Renew ChildNum cost: " << timer.getTimerMilliSec() << "ms" << std::endl;
	timer.update();

	hipFree(d_seedArr);
	hipFree(d_queue);
	hipFree(d_queueHead);
	hipFree(d_queueLock);
	hipFree(d_counter);
	hipFree(d_interSectValid);
	hipFree(d_seedRadiusMat);
	free(queue);
	free(qSize);
	free(counter);

}



//һ�������������ģ��
//a template for checking if two arrays identical
template<typename T>
void crosscheck(const T* d_arr1, const T* d_arr2, int arrSize)
{
	T* res1 = (T*)malloc(sizeof(T) * arrSize);
	T* res2 = (T*)malloc(sizeof(T) * arrSize);
	hipMemcpy(res1, d_arr1, sizeof(T) * arrSize, hipMemcpyDeviceToHost);
	hipMemcpy(res2, d_arr2, sizeof(T) * arrSize, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	hipError_t errorCheck;
	errorCheck = hipGetLastError();
	if (errorCheck != hipSuccess) {
		std::cerr << "In crosscheck: " << hipGetErrorString(errorCheck) << std::endl;
		system("pause");
		return;
	}


	int errorCount = 0;
	for (int i = 0; i < arrSize; i++)
	{
		if (res1[i] != res2[i])
		{
			printf("id: %d, res1: %d, res2: %d\n", i, res1[i], res2[i]);
			errorCount++;
		}
		if (errorCount > 500)
			break;
	}
	free(res1);
	free(res2);
	if (errorCount == 0)
		std::cerr << "Crosscheck is clear!" << std::endl;
}
