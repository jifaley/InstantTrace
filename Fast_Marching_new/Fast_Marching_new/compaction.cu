#include "hip/hip_runtime.h"
#include "compaction.h"
#include "TimerClock.hpp"

template<typename T>
struct is_non_zero {
	__host__ __device__
		bool operator()(T x) const
	{
		return x != 0;
	}
};

template<typename T>
struct is_zero {
	__host__ __device__
		bool operator()(T x) const
	{
		return x == 0;
	}
};


using thrust::placeholders::_1;

__global__
void getCompressMap(int* d_compress, int* d_decompress, uchar* d_imagePtr, uchar* d_imagePtr_compact, int newSize)
{
	int smallIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (smallIdx >= newSize) return;
	int fullIdx = d_decompress[smallIdx];

	d_compress[fullIdx] = smallIdx;
	d_imagePtr_compact[smallIdx] = d_imagePtr[fullIdx];
}


/*
������compactImage
���ܣ�ѹ��ԭͼ��ȥ����0���֡� 
�����d_compactedImagePtr(ѹ�����ͼ)��d_compress (ԭͼ->ѹ��ͼӳ��)��d_decompress(ѹ��ͼ->ԭͼӳ�䣩
˼·�����Ƚ��������غ����±��Ϊtuple��������(0,value0), (1, value1), (2,value2)....
������value< 0�Ĳ���ɾ����ʣ���tuple��Ϊ: (id0, value_id0), (id1, value_id1)...
��ô,ʣ���valueֵ��Ϊѹ�����ͼ��ʣ���id��Ϊѹ�����ֵ��Ӧ��ԭͼ�е��±ꡣ
ʵ�֣�ʹ��thrust���copy_if ���� remove_if ����
*/

void compactImage(uchar* d_imagePtr, uchar* &d_imagePtr_compact, int* &d_compress, int* &d_decompress, int width, int height, int slice, int& newSize)
{
	TimerClock timer;
	timer.update();

	hipError_t errorCheck;
	hipMalloc(&d_compress, sizeof(int) * width * height * slice);
	int* d_sequence = d_compress; //ԭ�����������顣Ϊ�˽�ʡ��������ʱ����ͬһ��ռ�

	//������50ms���ҵ�ͬ��ʱ�䣨��ʹȥ��cuDeiveSyncronize()��hipMemset()Ҳ��ǿ��ͬ����
	hipDeviceSynchronize();
	std::cerr << "stream compaction preprocess cost: " << timer.getTimerMilliSec() << "ms" << std::endl;
	timer.update();


	//����copy_if��d_sequence�����µ���ԭʼ�����ݷ�0ֵ���±ꡣ�ò�����stable�ġ� newSize��Ϊ��0ֵ�ĸ�����
	try
	{
		int* d_copy_end = thrust::copy_if(thrust::device, thrust::make_counting_iterator<int>(0), thrust::make_counting_iterator<int>(width * height * slice), d_imagePtr, d_sequence, _1 != 0);
		newSize = d_copy_end - d_sequence;
	}
	catch (thrust::system_error error)
	{
		std::cerr << std::string(error.what()) << std::endl;
	}

	hipMalloc(&d_decompress, sizeof(int) * newSize);
	hipMalloc(&d_imagePtr_compact, sizeof(uchar) * newSize);
	hipMemcpy(d_decompress, d_sequence, sizeof(int) * newSize, hipMemcpyDeviceToDevice);
	hipMemset(d_compress, 0xff, sizeof(int) * width * height * slice);

	//�����Ӧ��ӳ��
	getCompressMap << < (newSize - 1) / 256 + 1, 256 >> > (d_compress, d_decompress, d_imagePtr, d_imagePtr_compact, newSize);

	errorCheck = hipGetLastError();
	if (errorCheck != hipSuccess) {
		std::cerr << "Duing copyif " << hipGetErrorString(errorCheck) << std::endl;
		system("pause");
		return;
	}
	//�������㣬����copy_if ��getMap()��ʵ�ʺ�ʱԼ20ms����������50ms��ͬ���������ۡ�
}

struct isValid_functor {

	const uchar threshold;

	isValid_functor(uchar _th) : threshold(_th) {}
	__host__ __device__
		bool operator()(const uchar& x) const
	{
		return x >= threshold;
	}
};


__global__
void centerProcess(int* d_sequence, int* d_decompress, int maxSeedNum, int width, int height, int slice)
{
	__shared__ int sumX, sumY, sumZ;
	__shared__ int minDist;
	__shared__ int minPos;


	int tid = threadIdx.x;
	if (tid >= maxSeedNum) return;

	int smallIdx = d_sequence[tid];
	int fullIdx = d_decompress[smallIdx];

	int z = fullIdx / (width * height);
	int y = fullIdx % (width * height) / width;
	int x = fullIdx % width;

	atomicAdd(&sumZ, z);
	atomicAdd(&sumY, y);
	atomicAdd(&sumX, x);

	__syncthreads();

	if (tid == 0)
	{
		sumX = sumX / maxSeedNum;
		sumY = sumY / maxSeedNum;
		sumZ = sumZ / maxSeedNum;
		minDist = 2147483647;
		d_sequence[0] = 2147483647;
	}

	__syncthreads();

	int dist = sqrtf((sumZ - z) * (sumZ - z) + (sumY - y) * (sumY - y) + (sumX - x) * (sumX - x));

	atomicMin(&minDist, dist);

	__syncthreads();

	if (minDist == dist)
	{
		atomicMin(&d_sequence[0], fullIdx);
	}
}

void getCenterPos(int* d_compress, int* d_decompress, uchar* d_radiusMat_compact, int width, int height, int slice, int newSize, int&maxPos, int& maxRadius)
{
	thrust::device_ptr<uchar> d_ptr(d_radiusMat_compact);
	thrust::device_ptr<uchar> iter = thrust::max_element(d_ptr, d_ptr + newSize);
	maxRadius = *iter;

	printf("Init maxRadius: %d\n", maxRadius);
	
	int* d_sequence;
	hipMalloc(&d_sequence, sizeof(int) * newSize);

	uchar thresholdRadius = MAX(maxRadius * 4 / 5, maxRadius - 5);

	int* d_copy_end = thrust::copy_if(thrust::device, thrust::make_counting_iterator<int>(0), thrust::make_counting_iterator<int>(newSize), d_radiusMat_compact, d_sequence,isValid_functor(thresholdRadius));
	int maxSeedNum = d_copy_end - d_sequence;

	maxSeedNum = MIN(maxSeedNum, 512);

	centerProcess << <1, maxSeedNum >> > (d_sequence, d_decompress, maxSeedNum, width, height, slice);

	thrust::device_ptr<int> dp(d_sequence);

	maxPos = *dp;
	hipFree(d_sequence);
}